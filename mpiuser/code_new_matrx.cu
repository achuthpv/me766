#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#define ThreadSize 16
__global__ void MatMulKernel( int *dD, int *dE, int *dF, int N ) {
	int Fvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ( row < (N/2) && col < (N/2) ) {
	

	for ( int i=0; i<(N); i++ ) {
          Fvalue+= dD[(row)*(N)+i] * dE[i*(N/2)+col];
	}

	dF[row*(N/2)+col]=Fvalue;
	}
}

extern "C" int CUDA_stuff(int *D, int *E, int *F, int N )
{

	int i, j;
	
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
	hipError_t err;
//	printf ( "N = %d\n", N );
	
	int *dD, *dE, *dF;
//	if(my_rank==0){
/* printf("E Matrix is");
        
        for(i=0;i<(N/2);i++){
             for(j=0;j<(N/2);j++){
                  printf("~%d ",E[i*(N/2)+j]);
                }
                  printf("\n");
                }

printf("D Matrix is");
	        for(i=0;i<(N/2);i++){
                     for(j=0;j<(N/2);j++){
                        printf("~%d ",D[i*(N/2)+j]);
                        }
                          printf("\n");
                        }
//}*/
// Allocate the memory on the GPU
   hipEventRecord(start,0);
	
	err = hipMalloc ( (void**) &dD, (N/2)*(N)*sizeof(int));
//	printf ( "CUDA malloc dD: %s\n", cudaGetErrorString(err));
	err = hipMalloc ((void**) &dE, (N/2)*(N)*sizeof(int));
//	printf ( "CUDA malloc dE: %s\n", cudaGetErrorString(err));
	err = hipMalloc ((void**) &dF, (N/2)*(N/2)*sizeof(int));
//	printf ( "CUDA malloc dF: %s\n", cudaGetErrorString(err));
	
	// Copy the memory to the GPU.
	err = hipMemcpy(dD, D, (N/2)*(N)*sizeof(int), hipMemcpyHostToDevice );
//	printf ( "Copy D to device: %s\n", cudaGetErrorString(err));
	err = hipMemcpy(dE, E, (N/2)*(N)*sizeof(int), hipMemcpyHostToDevice );
//	printf ( "Copy E to device: %s\n", cudaGetErrorString(err));
	err = hipMemcpy(dF, F, (N/2)*(N/2)*sizeof(int), hipMemcpyHostToDevice );
//	printf ( "Copy F to device: %s\n", cudaGetErrorString(err));
/*         
        for(i=0;i<(N/2);i++){
          for(j=0;j<(N/2);j++){
         printf("%d ",D[i*(N/2)+j]);
}
printf("\n");
}*/
	dim3 dimBlock(ThreadSize,ThreadSize,1);
	dim3 dimGrid(((N/ThreadSize)+1),((N/ThreadSize)+1),1);
	
	// Perform the operation on the GPU
	
	MatMulKernel <<< dimGrid, dimBlock >>> (dD, dE, dF, N);
	
	// Copy back the results from the GPU to the CPU
	
	err = hipMemcpy ( F, dF, (N/2)*(N/2)*sizeof(int), hipMemcpyDeviceToHost );
//	printf ( "Copy F off of device %s\n", cudaGetErrorString(err) );

		

hipEventRecord(stop,0);
hipEventSynchronize(stop);
float GPUelapsed;
hipEventElapsedTime(&GPUelapsed,start,stop);

//printf("\n\nGPU Elapsed time:%f\n\n",GPUelapsed);




/*	for ( i=0; i<N/2; i++ ) {
		for ( j=0; j<N/2; j++ ) {
			printf ( "%3d ", F[i*N/2+j] );
		}
		printf ( "\n" );
	}
	printf ( "\n" );
*/
	printf ( "Success!\n" );
	
	return(0);
}



