#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

__global__ void __multiply__(float *d_A,float *d_B,float *d_C,int N){

   int row=threadIdx.y+blockIdx.y*blockDim.y;
   int col=threadIdx.x+blockIdx.x*blockDim.y;


if(col<N && row<N){

            float sum=0;
        for(int i=0;i<N;i++){
       sum +=d_A[row*N+i]*d_B[i*N+col];
     
 }       
    d_C[row*N+col]=sum;
}

}

extern "C" void launch_multiply(float *d_A,float *d_B,float *d_C,int N)
{
  ,

}
