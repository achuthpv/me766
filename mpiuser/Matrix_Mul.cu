#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "mpi.h"
#define N 100
//---------------------------------------------------------------------------------------------------------------

#define BLOCKSIZE 16

//---------------------------------------------------------------------------------------------------------------

int  IntializingVectors(float **, float **);
int CheckDevice(int );

//----------------------------------------------------------------------------------------------------------------

//Pragma routine to report the detail of cuda error

#define CUDA_SAFE_CALL(call)                                                         \
            do{                                                                      \
                 hipError_t err = call;                                             \
                 if(err != hipSuccess)                                              \
                 {                                                                   \
                        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                         __FILE__, __LINE__, hipGetErrorString( err) );             \
                         exit(1);                                                    \
                 }                                                                   \
               } while (0)                                                           \

//------------------------------------------------------------------------------------------------------------

//Kernel that performs Vector Vector Dot Product
__global__ void matrixProduct(float *Vector1,float *Vector2,float *DeviceTempVector)
{
        int tidx = threadIdx.x+blockIdx.x*blockDim.x;
        int tidy = threadIdx.y+blockIdx.y*blockDim.y;
       // int ThreadIndex = (ThreadDim * tidx) + tidy;
       // int MaxNumThread = ThreadDim * ThreadDim;
       // int ThreadColumnIndex,RowNum,pass = 0;
       // *Solution = 0.0;
        float sum ;
         int  i;
        sum=0;
       if(tidx<N && tidy<N )
        {
                for(i=0;i<N;i++){
                sum +=(Vector1[tidy*N+i]) * (Vector2[i*N+tidx]) ;
                }
             DeviceTempVector[tidy*N+tidx]=sum;
        }

}
//-------------------------------------------------------------------------------------------------------------------------------------

int main(int argc, char **argv)
{
  //            int N;
//  printf("enter The size: ");
//scanf("%d",&N);
	int MyRank, NumberOfProcessors, VectorSize, ScatterSize;
	float *VectorA, *VectorB,*VectorC;
	float *MyVectorA, *MyVectorB,*MyVectorC;
	float *DeviceMyVectorA, *DeviceMyVectorB, *DeviceTempVector;
	float *NodeSum; 
	float *Result;
	int DeviceStatus, Index,Index1, Root = 0, Status = 1;

	//Intilaizing the MPI
	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &MyRank);
	MPI_Comm_size(MPI_COMM_WORLD, &NumberOfProcessors);

	//Checking if valid number of arguements have been passed
        if(argc != 3)
	{
		if(MyRank == Root)
			printf("Usage:< mpirun >< -n >< Number of Processors >< ./Program Name >< Vector Size > \n");	
		MPI_Finalize();
		exit(0);
	}	

	//Getting Vector Size program arguements
	VectorSize = N*N;
	
	//Checking if Vector size is less than the Total number of processors
	if(VectorSize < NumberOfProcessors)
	{
		MPI_Finalize();
		if(MyRank == Root)
		{
			printf("Vector Size should be more than Number of processors \n");
			exit(-1);
		}
		exit(-1);
	}
	
	//Checking if data can be distributed evenly to all nodes 
	if(VectorSize % NumberOfProcessors != 0)
	{
		MPI_Finalize();
		if(MyRank == Root)
			printf("Vectos can not be distributed evenly among all processors \n");
		exit(-1);
	}

	//Root node intializes the VectorA and VectorB
       if(MyRank == Root)
                Status = IntializingVectors(&VectorA, &VectorB);

        MPI_Bcast(&Status, 1, MPI_INT, Root, MPI_COMM_WORLD);

        //Checking if status returned by the function IntializingVectors is zero
        if(Status == 0)
        {
                if(MyRank == Root)
                        printf("I am processor %d and the memory is not available for the varilable on the host \n",MyRank);
                MPI_Finalize();
                exit(-1);
        }
		
	//Calculating the Scatter size
	ScatterSize = VectorSize / NumberOfProcessors;
	
	//Allocating memory on the host by all the nodes
	MyVectorA = (float *)malloc(ScatterSize * sizeof(float));
	MyVectorB = (float *)malloc(ScatterSize * sizeof(float));
	NodeSum = (float *)malloc(ScatterSize*sizeof(float));
        Result = (float *)malloc(ScatterSize*sizeof(float));

	//Distributing the VectorA and VectorB to all the nodes
	MPI_Scatter(VectorA, ScatterSize, MPI_FLOAT, MyVectorA, ScatterSize, MPI_FLOAT, Root, MPI_COMM_WORLD);
	MPI_Scatter(VectorB, ScatterSize, MPI_FLOAT, MyVectorB, ScatterSize, MPI_FLOAT, Root, MPI_COMM_WORLD);
        MPI_Scatter(VectorC, ScatterSize, MPI_FLOAT, MyVectorC, ScatterSize, MPI_FLOAT, Root, MPI_COMM_WORLD);
	DeviceStatus = CheckDevice(MyRank);

	if(DeviceStatus == 0)
        {
                printf("Processor with rank %d doing partial product of two Matrix  on CPU \n",MyRank);
                for(Index = 0; Index < ScatterSize; Index++)
                   for(Index1=0;Index < ScatterSize;Index1++)
                           (*NodeSum) += (MyVectorA[Index*ScatterSize+Index1] * MyVectorB[Index1*ScatterSize+Index]);
        }
	else
	{
		//allocating memory on the Device memory
		CUDA_SAFE_CALL( hipMalloc( (void **)&DeviceMyVectorA, ScatterSize * sizeof(float) ) );
		CUDA_SAFE_CALL( hipMalloc( (void **)&DeviceMyVectorB, ScatterSize * sizeof(float) ) );
		CUDA_SAFE_CALL( hipMalloc( (void **)&DeviceTempVector, ScatterSize * sizeof(float) ) );
		//CUDA_SAFE_CALL( hipMalloc( (void **)&DeviceNodeSum, ScatterSize*sizeof(float) ) );

		//Copying data from Host to device memory
		CUDA_SAFE_CALL( hipMemcpy( (void *)DeviceMyVectorA, (void *)MyVectorA, ScatterSize * sizeof(float), hipMemcpyHostToDevice ) );
		CUDA_SAFE_CALL( hipMemcpy( (void *)DeviceMyVectorB, (void *)MyVectorB, ScatterSize * sizeof(float), hipMemcpyHostToDevice) );

		//Defining Thread Grid and Thread Block
		dim3 DimGrid(((N+1)/BLOCKSIZE),((N+1)/BLOCKSIZE));
		dim3 DimBlock(BLOCKSIZE, BLOCKSIZE);
	
		//Calling the kernel which performs Vector Vector Product
		matrixProduct<<<DimGrid, DimBlock>>>(DeviceMyVectorA, DeviceMyVectorB, DeviceTempVector);
	
		//Copying the value of the node sum from the Device to the Host
		CUDA_SAFE_CALL( hipMemcpy((void *)NodeSum, (void *)DeviceTempVector, sizeof(float), hipMemcpyDeviceToHost) ); 	
	}
	
	MPI_Barrier(MPI_COMM_WORLD);

	//Adding the NodeSum value from all the nodes to get the final product value 
	MPI_Reduce(NodeSum, &Result, 1, MPI_FLOAT, MPI_SUM, Root, MPI_COMM_WORLD);

	//Root node printing the product of given two vectors 
	if(MyRank == Root)
	{
		//printf("The product of the given two Matrix is %f \n", Result);
		//Freeing the Vectors allocated by root node
		free(VectorA);
		free(VectorB);
	}

	//Freeing the Host Memory
	free(MyVectorA);
	free(MyVectorB);
	free(NodeSum);

	//Freeing  the Device Memory
	CUDA_SAFE_CALL( hipFree(DeviceMyVectorA) );
	CUDA_SAFE_CALL( hipFree(DeviceMyVectorB) );
	CUDA_SAFE_CALL( hipFree(DeviceTempVector) );
	//CUDA_SAFE_CALL( hipFree(DeviceNodeSum) );	
		
	MPI_Finalize();

	return(0);

}//End of Main function
//------------------------------------------------------------------------------------------------------------------------------------

int  IntializingVectors(float **VectorA, float **VectorB)
{
	float *TempVectorA, *TempVectorB,*TempVectorC;
	int Index,index1, Status = 1;

	TempVectorA = (float *)malloc(N*N * sizeof(float));
	if(TempVectorA == NULL)
		Status = 0;
	
	TempVectorB = (float *)malloc(N*N * sizeof(float));
	if(TempVectorB == NULL)	
		Status = 0;
		TempVectorC= (float *)malloc(N*N * sizeof(float));
	if(TempVectorC == NULL)	
		Status = 0;
	
	for(Index = 0; Index < N; Index++)
              for(index1=0;index1<N;index1++)
	{
		TempVectorA[Index*N+index1] = Index * index1;
	
	TempVectorB[Index*N+index1] = Index + index1;
	TempVectorC[Index*N+index1] = 0;

	}
	
	*VectorA = TempVectorA;
	*VectorB = TempVectorB;
	
	return(Status);
}//End of function
//----------------------------------------------------------------------------------------------------------------------------------

int CheckDevice(int MyRank)
{
        int DeviceCount, Device;
        struct hipDeviceProp_t Properties;

        hipGetDeviceCount(&DeviceCount);
        if(DeviceCount >= 1)
        {
                hipGetDevice(&Device);
                hipGetDeviceProperties(&Properties, Device);
                printf("Processor with  rank %d has the Device by name %s and compuatation is doen on this device\n",MyRank, Properties.name);
        }

        return(DeviceCount);

}//End of function
//--------------------------------------------------------------------------------------------------------------


